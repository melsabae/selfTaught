
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <unistd.h>
#include <memory>
#include <algorithm>
#include <array>
#include <numeric>


//const std::size_t N = 1 << 10;
//const std::size_t N = 1 << 8; // any higher of a shift and this will fail for a N*N*N matrix
const std::size_t N = 1 << 6; // any higher of a shift and this will fail for a N*N matrix


__global__
// increment all elements in parallel
void vec_inc(float* const c, const std::size_t n)
{
    for (std::size_t x = threadIdx.x + (blockIdx.x * blockDim.x); x < n; x += (blockDim.x * gridDim.x))
    {
        for (std::size_t y = threadIdx.y + (blockIdx.y * blockDim.y); y < n; y += (blockDim.y * gridDim.y))
        {
            for (std::size_t z = threadIdx.z + (blockIdx.z * blockDim.z); z < n; z += (blockDim.z * gridDim.z))
            {
                ++ c[(x * n * n) + (y * n) + z];
            }
        }
    }
}


int main(void)
{
    // grids and blocks are topologically laid out similar to the problem
    // for 1D arrays, a grid size of (1,1,1) = 1D, and a block size of (N,1,1) = N, would be enough to fully cover the array
    const dim3 grid_size(1, 1, 1);
    const dim3 block_size(512, 1, 1);

    std::size_t error_count = 0; // track errors found
    float* d_a = nullptr; // handle on CUDA memory
    //std::shared_ptr<std::array<float, N * N * N>> h_mat(new std::array<float, N * N * N>()); // matrix we want to do grid-strides
    std::shared_ptr<std::array<float, N * N>> h_mat(new std::array<float, N * N>()); // matrix we want to do grid-strides

    std::iota(h_mat->begin(), h_mat->end(), 0);

    hipStream_t stream;

    if (nullptr == h_mat)
    {
        std::cout << __FUNCTION__ << " " << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipStreamCreate(&stream))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipMalloc(&d_a, h_mat->size() * sizeof(d_a[0])))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipMemcpyAsync(d_a, h_mat->data(), h_mat->size() * sizeof(d_a[0]), hipMemcpyHostToDevice, stream))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    vec_inc<<<grid_size, block_size, 0, stream>>>(d_a, N);

    if (hipSuccess != hipMemcpyAsync(h_mat->data(), d_a, h_mat->size() * sizeof(d_a[0]), hipMemcpyDeviceToHost, stream))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipStreamSynchronize(stream))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    for (std::size_t i = 0; i < h_mat->size(); ++i)
    {
        if ((*h_mat)[i] != i + 1)
        {
            ++ error_count;
        }
    }

    std::cout << error_count << "/" << h_mat->size() << std::endl << std::flush;

    hipFree(d_a);
    hipStreamDestroy(stream);
    return 0;
}

