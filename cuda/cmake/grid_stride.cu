
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <unistd.h>
#include <memory>
#include <algorithm>
#include <array>
#include <numeric>


//const std::size_t N = 1 << 10;
const std::size_t N = 1 << 8;


__global__
// c[x], c[y], c[z] are all incremented
// c is an n*n*n matrix
void vec_inc(float* const c, const std::size_t n)
{
    // it is much faster to treat a matrix as an array than as a matrix, at least in this kernel
    for (std::size_t x = threadIdx.x + (blockIdx.x * blockDim.x); x < n*n*n; x += (blockDim.x * gridDim.x))
    {
        ++c[x];
    }

    //for (std::size_t x = threadIdx.x + (blockIdx.x * blockDim.x); x < n; x += (blockDim.x * gridDim.x))
    //{
    //    for (std::size_t y = threadIdx.y + (blockIdx.y * blockDim.y); y < n; y += (blockDim.y * gridDim.y))
    //    {
    //        for (std::size_t z = threadIdx.z + (blockIdx.z * blockDim.z); z < n; z += (blockDim.z * gridDim.z))
    //        {
    //            ++ c[(x * n * n) + (y * n) + z];
    //        }
    //    }
    //}
}


int main(void)
{
    // grids and blocks are topologically laid out similar to the problem
    // for 1D arrays, a grid size of (1,1,1) = 1D, and a block size of (N,1,1) = N, would be enough to fully cover the array
    const dim3 grid_size(1, 1, 1);
    const dim3 block_size(1024, 1, 1);

    std::size_t error_count = 0; // track errors found
    float* d_a = nullptr; // handle on CUDA memory
    std::shared_ptr<std::array<float, N * N * N>> h_mat(new std::array<float, N * N * N>()); // matrix we want to do grid-strides

    std::iota(h_mat->begin(), h_mat->end(), 0);

    if (nullptr == h_mat)
    {
        std::cout << __FUNCTION__ << " " << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipMalloc(&d_a, h_mat->size() * sizeof(d_a[0])))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipMemcpyAsync(d_a, h_mat->data(), h_mat->size() * sizeof(d_a[0]), hipMemcpyHostToDevice))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    vec_inc<<<grid_size, block_size>>>(d_a, N);

    if (hipSuccess != hipMemcpyAsync(h_mat->data(), d_a, h_mat->size() * sizeof(d_a[0]), hipMemcpyDeviceToHost))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipDeviceSynchronize())
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    for (std::size_t i = 0; i < h_mat->size(); ++i)
    {
        if ((*h_mat)[i] != i + 1)
        {
            ++ error_count;
        }
    }

    std::cout << error_count << " " << h_mat->size() << std::endl << std::flush;

    hipFree(d_a);
    return 0;
}

