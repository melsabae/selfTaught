
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <unistd.h>
#include <memory>
#include <algorithm>
#include <array>
#include <numeric>


//const std::size_t N = 1 << 10;
const std::size_t N = 2;


__global__
// c[x], c[y], c[z] are all incremented
// c is an n*n*n matrix
void vec_inc(float* const c, const std::size_t n)
{
    for (std::size_t x = threadIdx.x + (blockIdx.x * blockDim.x); x < n; x += (blockDim.x * gridDim.x))
    {
        for (std::size_t y = threadIdx.y + (blockIdx.y * blockDim.y); y < n; y += (blockDim.y * gridDim.y))
        {
            for (std::size_t z = threadIdx.z + (blockIdx.z * blockDim.z); z < n; z += (blockDim.z * gridDim.z))
            {
                ++ c[n * (x + y + z)];
            }
        }
    }
}


int main(void)
{
    // grids and blocks are topologically laid out similar to the problem
    // for 1D arrays, a grid size of (1,1,1) = 1D, and a block size of (N,1,1) = N, would be enough to fully cover the array
    const dim3 grid_size(1, 1, 1);
    const dim3 block_size(256, 2, 2);

    std::size_t error_count = 0;

    std::shared_ptr<std::array<float, N * N * N>> h_mat(new std::array<float, N * N * N>());
    std::iota(h_mat->begin(), h_mat->end(), 0);

    for (auto v: *h_mat)
    {
        std::cout << v << std::endl;
    }

    float* d_a = nullptr;
    hipStream_t stream;

    if (nullptr == h_mat)
    {
        std::cout << __FUNCTION__ << " " << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipStreamCreate(&stream))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipMalloc(&d_a, h_mat->size() * sizeof(d_a[0])))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipMemcpyAsync(d_a, h_mat->data(), h_mat->size() * sizeof(d_a[0]), hipMemcpyHostToDevice, stream))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    vec_inc<<<grid_size, block_size, 0, stream>>>(d_a, N);

    if (hipSuccess != hipMemcpyAsync(h_mat->data(), d_a, h_mat->size() * sizeof(d_a[0]), hipMemcpyDeviceToHost, stream))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipStreamSynchronize(stream))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    for (std::size_t i = 0; i < h_mat->size(); ++i)
    {
        if ((*h_mat)[i] != i + 1)
        {
            ++ error_count;
        }
    }

    for (auto v: *h_mat)
    {
        std::cout << v << std::endl;
    }

    std::cout << error_count << " " << h_mat->size() << std::endl << std::flush;

    hipFree(d_a);
    hipStreamDestroy(stream);
    return 0;
}

