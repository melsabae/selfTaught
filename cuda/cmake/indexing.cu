
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <unistd.h>
#include <algorithm>
#include <numeric>
#include <vector>
#include <cassert>


constexpr size_t side = 2;
constexpr size_t num_el = side * side * side;


__global__
void f(int32_t* data, const size_t x)
{
    // assume data is being horizontally striped by blocks
    for (size_t i = (blockIdx.x * blockDim.x) + threadIdx.x; i < x; i += blockDim.x * gridDim.x)
    {
        data[i] = i + 2;
    }
}


__global__
void g(int32_t* data, const size_t x, const size_t y)
{
    // assume data is being horizontally striped by blocks
    for (size_t i = (blockIdx.x * blockDim.x) + threadIdx.x; i < x; i += blockDim.x * gridDim.x)
    {
        data[y * threadIdx.y + i] = (threadIdx.y * 2 + threadIdx.x + 1) * 20;
    }
}


__global__
void h(int32_t* data, const size_t x, const size_t y, const size_t z)
{
    // assume data is being horizontally striped by blocks
    for (size_t i = (blockIdx.x * blockDim.x) + threadIdx.x; i < x; i += blockDim.x * gridDim.x)
    {
        data[z * (y * threadIdx.y + x * threadIdx.x) + (y * threadIdx.y) + i] = (threadIdx.z * 4 + threadIdx.y * 2 + threadIdx.x + 1) * 20;
    }
}


void reset(int32_t* data)
{
    memset(data, -1, sizeof(*data) * num_el * 3);

    for (size_t i = 0; i < 3 * num_el; ++ i)
    {
        if (i >= num_el and i < 2 * num_el)
        {
            data[i] = i;
        }
    }
}


void print(int32_t* data)
{
    for (size_t i = 0; i < 3 * num_el; ++ i)
    {
        if (data[i] == -1)
        {
            std::cout << ",";
        }
        else
        {
            std::cout << data[i] << ",";
        }
    }
    std::cout << std::endl;
}


size_t num_diff(int32_t* data)
{
    size_t n = 0;

    for (size_t i = 0; i < 3 * num_el; ++ i)
    {
        if (i >= num_el and i < 2 * num_el)
        {
            if (data[i] != i)
            {
                n += 1;
            }
        }
        else
        {
            if (data[i] != -1)
            {
                n += 1;
            }
        }
    }

    return n;
}


void test_f(int32_t* data, hipStream_t& stream)
{
    constexpr dim3 grid_size { 1, 1, 1 };
    constexpr dim3 block_size { 1024, 1, 1 };

    std::cout << ">>> test_f" << std::endl;
    reset(data);
    print(data);

    f<<<grid_size, block_size, 0, stream>>>(data + num_el, side);

    if (hipSuccess != hipStreamSynchronize(stream))
    {
        std::cerr << "bailing " << __LINE__ << std::endl;
        exit(__LINE__);
    }

    print(data);
    std::cout << "<<< test_f " << num_diff(data) << std::endl << std::endl;
}


void test_g(int32_t* data, hipStream_t& stream)
{
    constexpr dim3 grid_size { 1, 1, 1 };
    constexpr dim3 block_size { 1024 / side, side, 1 };

    std::cout << ">>> test_g" << std::endl;
    reset(data);
    print(data);

    g<<<grid_size, block_size, 0, stream>>>(data + num_el, side, side);

    if (hipSuccess != hipStreamSynchronize(stream))
    {
        std::cerr << "bailing " << __LINE__ << std::endl;
        exit(__LINE__);
    }

    print(data);
    std::cout << "<<< test_g " << num_diff(data) << std::endl << std::endl;
}


void test_h(int32_t* data, hipStream_t& stream)
{
    constexpr dim3 grid_size { 1, 1, 1 };
    constexpr dim3 block_size { 1024 / (side * side), side, side };

    std::cout << ">>> test_h" << std::endl;
    reset(data);
    print(data);

    h<<<grid_size, block_size, 0, stream>>>(data + num_el, side, side, side);

    if (hipSuccess != hipStreamSynchronize(stream))
    {
        std::cerr << "bailing " << __LINE__ << std::endl;
        exit(__LINE__);
    }

    print(data);
    std::cout << "<<< test_h " << num_diff(data) << std::endl << std::endl;
}


int main(void)
{
    static_assert(1 < side);
    static_assert(1 < num_el && num_el <= 1024);

    hipStream_t stream;
    int32_t* data_f = nullptr;
    int32_t* data_g = nullptr;
    int32_t* data_h = nullptr;

    if (
           // pad the area of data with just some canary zeros
           hipSuccess != hipMallocManaged(&data_f, 3 * num_el * sizeof(*data_f))
        or hipSuccess != hipMallocManaged(&data_g, 3 * num_el * sizeof(*data_g))
        or hipSuccess != hipMallocManaged(&data_h, 3 * num_el * sizeof(*data_h))
        or hipSuccess != hipStreamCreate(&stream)
       )
    {
        std::cerr << "bailing " << __LINE__ << std::endl;
        return -__LINE__;
    }

    test_f(data_f, stream);
    test_g(data_g, stream);
    test_h(data_h, stream);

    hipFree(data_f);
    hipFree(data_g);
    hipFree(data_h);
    hipStreamDestroy(stream);
    return 0;
}

