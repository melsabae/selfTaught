
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <unistd.h>
#include <memory>
#include <array>
#include <algorithm>
#include <vector>


const std::size_t NUMBER_STREAMS = 1;
const std::size_t N = 1 << 20;


struct test_struct
{
    hipStream_t stream;

    std::unique_ptr<std::array<float, N>> h_a;
    std::unique_ptr<std::array<float, N>> h_b;
    std::unique_ptr<std::array<float, N>> h_c;

    float* d_a;
    float* d_b;
    float* d_c;

    std::size_t error_count;
};


typedef struct test_struct test_struct_t;


int make_test_struct(test_struct_t& t)
{
    t.h_a.reset(new std::array<float, N>);
    t.h_b.reset(new std::array<float, N>);
    t.h_c.reset(new std::array<float, N>);

    if (nullptr == t.h_a)
    {
        std::cout << __FUNCTION__ << " " << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (nullptr == t.h_a)
    {
        std::cout << __FUNCTION__ << " " << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (nullptr == t.h_a)
    {
        std::cout << __FUNCTION__ << " " << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipStreamCreate(&t.stream))
    {
        std::cout << __FUNCTION__ << " " << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipMalloc(&t.d_a, N * sizeof(t.d_a[0])))
    {
        std::cout << __FUNCTION__ << " " << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipMalloc(&t.d_b, N * sizeof(t.d_b[0])))
    {
        std::cout << __FUNCTION__ << " " << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipMalloc(&t.d_c, N * sizeof(t.d_c[0])))
    {
        std::cout << __FUNCTION__ << " " << __LINE__ << std::endl;
        return -__LINE__;
    }

    std::fill(t.h_a->begin(), t.h_a->end(), 1.0);
    std::fill(t.h_b->begin(), t.h_b->end(), 2.0);

    t.error_count = 0;

    return 0;
}


int destroy_test_struct(test_struct_t& t)
{
    if (hipSuccess != hipFree(t.d_a))
    {
        std::cout << __FUNCTION__ << " " << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipFree(t.d_b))
    {
        std::cout << __FUNCTION__ << " " << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipFree(t.d_c))
    {
        std::cout << __FUNCTION__ << " " << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipStreamDestroy(t.stream))
    {
        std::cout << __FUNCTION__ << " " << __LINE__ << std::endl;
        return -__LINE__;
    }

    return 0;
}


__global__
void vec_add(float* const c, const float* const a, const float* const b, const std::size_t n)
{
    for (std::size_t i = threadIdx.x + (blockIdx.x * blockDim.x); i < n; i += (blockDim.x * gridDim.x))
    {
        c[i] = a[i] + b[i] + 1;
    }
}


int main(void)
{
    const dim3 grid_size (1, 1, 1);
    const dim3 block_size (1024, 1, 1);

    std::array<test_struct_t, NUMBER_STREAMS> streams;

    std::for_each(streams.begin(), streams.end(), make_test_struct);

    for (test_struct_t& t: streams)
    {
        if (hipSuccess != hipMemcpyAsync(t.d_a, t.h_a->data(), N * sizeof(t.d_a[0]), hipMemcpyHostToDevice, t.stream))
        {
            std::cout << __FUNCTION__ << " " << __LINE__ << std::endl;
            return -__LINE__;
        }

        if (hipSuccess != hipMemcpyAsync(t.d_b, t.h_b->data(), N * sizeof(t.d_b[0]), hipMemcpyHostToDevice, t.stream))
        {
            std::cout << __FUNCTION__ << " " << __LINE__ << std::endl;
            return -__LINE__;
        }

        vec_add<<<grid_size, block_size, 0, t.stream>>>(t.d_c, t.d_a, t.d_b, N);

        if (hipSuccess != hipMemcpyAsync(t.h_c->data(), t.d_c, N * sizeof(t.d_c[0]), hipMemcpyDeviceToHost, t.stream))
        {
            std::cout << __FUNCTION__ << " " << __LINE__ << std::endl;
            return -__LINE__;
        }
    }

    for (test_struct_t& t: streams)
    {
        if (hipSuccess != hipStreamSynchronize(t.stream))
        {
            std::cout << __FUNCTION__ << " " << __LINE__ << std::endl;
            return -__LINE__;
        }
    }

    for (test_struct_t& t: streams)
    {
        for (std::size_t i = 0; i < N; ++ i)
        {
            if (4 != t.h_c->at(i))
            {
                ++ t.error_count;
            }
        }


        if (0 != t.error_count)
        {
            std::cout << t.error_count << std::endl << std::flush;
        }
    }

    std::for_each(streams.begin(), streams.end(), destroy_test_struct);

    return 0;
}

