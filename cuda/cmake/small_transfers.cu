
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <unistd.h>
#include <memory>
#include <algorithm>
#include <array>
#include <numeric>



// the output incorrectly says that the data mismatches, but it appears to be an issue with doubles
// changing all types to integral types showed 0 issues, which for small N, was also hand verified
// moving to double reduced the error rate


const std::size_t N = 1 << 3;
const double h_m = 7.2;
const double h_b = 3.4;


struct linear_params
{
    double m;
    double b;
};


typedef struct linear_params linear_params_t;


__global__
void linear(double* y, const std::size_t n, const double* x, const double m, const double b)
{
    for (std::size_t i = threadIdx.x + (blockIdx.x * blockDim.x); i < n; i += (blockDim.x * gridDim.x))
    {
        y[i] = m * x[i] + b;
    }
}


int main(void)
{
    const dim3 grid_size(1, 1, 1);
    const dim3 block_size(1024, 1, 1);

    std::unique_ptr<std::array<double, N>> h_x (new std::array<double, N>());
    std::unique_ptr<std::array<double, N>> h_y (new std::array<double, N>());
    double* d_y = NULL;
    double* d_x = NULL;
    hipStream_t stream;

    std::iota(h_x->begin(), h_x->end(), 0);

    if (hipSuccess != hipStreamCreate(&stream))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipHostMalloc(&d_y, sizeof(d_y[0]) * h_y->size(), hipHostMallocDefault))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipHostMalloc(&d_x, sizeof(d_x[0]) * h_x->size(), hipHostMallocDefault))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipMemcpyAsync(d_x, h_x->data(), sizeof(d_x[0]) * N, hipMemcpyHostToDevice, stream))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    linear<<<grid_size, block_size, 0, stream>>>(d_y, N, d_x, h_m, h_b);

    if (hipSuccess != hipMemcpyAsync(h_y->data(), d_y, sizeof(d_y[0]) * N, hipMemcpyDeviceToHost, stream))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipStreamSynchronize(stream))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    std::size_t error_count = 0;

    for (std::size_t i = 0; i < h_x->size(); ++i)
    {
        if (((h_m * (*h_x)[i]) + h_b) != (*h_y)[i])
        {
            // because the GPU and CPU compute differences in floats, output what the difference is
            // it should be very small
            std::cout << (((h_m * (*h_x)[i]) + h_b) - (*h_y)[i]) << std::endl;
            ++ error_count;
        }
    }

    //for (std::size_t i = 0; i < h_x->size(); ++i)
    //{
    //    std::cout << h_y(i) << " = (" << m << ")" << h_x(i) << " + " << b << std::endl;
    //}

    //for (std::size_t i = 0; i < h_x->size(); ++i)
    //{
    //    std::cout << h_y(i) << "," << m << "," << h_x(i) << "," << b << std::endl;
    //}

    std::cout << error_count << "/" <<  N << std::endl << std::flush;

    hipHostFree(d_x);
    hipHostFree(d_y);
    hipStreamDestroy(stream);
    return 0;
}

