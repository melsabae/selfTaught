
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <unistd.h>
#include <memory>
#include <algorithm>
#include <vector>

const std::size_t N = 1 << 20;

__global__
void vec_add(float* const c, const float* const a, const float* const b, const std::size_t n)
{
    for (std::size_t i = threadIdx.x + (blockIdx.x * blockDim.x); i < n; i += (blockDim.x * gridDim.x))
    {
        c[i] = a[i] + b[i];
    }
}

int main(void)
{
    // grids and blocks are topologically laid out similar to the problem
    // for 1D arrays, a grid size of (1,1,1) = 1D, and a block size of (N,1,1) = N, would be enough to fully cover the array
    const dim3 grid_size(1, 1, 1);
    const dim3 block_size(1024, 1, 1);

    std::vector<float> h_a(N);
    std::vector<float> h_b(N);
    std::vector<float> h_c(N);
    std::size_t error_count = 0;

    float* d_a = NULL;
    float* d_b = NULL;
    float* d_c = NULL;

    hipStream_t stream;

    std::fill(h_a.begin(), h_a.end(), 1.0);
    std::fill(h_b.begin(), h_b.end(), 2.0);

    if (hipSuccess != hipStreamCreate(&stream))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipMalloc(&d_a, N * sizeof(h_a[0])))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipMalloc(&d_b, N * sizeof(h_b[0])))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipMalloc(&d_c, N * sizeof(h_c[0])))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipMemcpyAsync(d_a, h_a.data(), N * sizeof(h_a[0]), hipMemcpyHostToDevice, stream))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipMemcpyAsync(d_b, h_b.data(), N * sizeof(h_b[0]), hipMemcpyHostToDevice, stream))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    vec_add<<<grid_size, block_size, 0, stream>>>(d_c, d_a, d_b, N);

    if (hipSuccess != hipMemcpyAsync(h_c.data(), d_c, N * sizeof(h_c[0]), hipMemcpyDeviceToHost, stream))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipStreamSynchronize(stream))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    for (std::size_t i = 0; i < N; ++ i)
    {
        if (h_a[i] + h_b[i] != h_c[i])
        {
            //std::cout << i << " " << h_c[i] << std::endl;
            ++ error_count;
        }
    }

    std::cout << error_count << " " << 100.0 * (static_cast<double>(error_count) / N) << "% mismatched" << std::endl << std::flush;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipStreamDestroy(stream);
    return 0;
}

