
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <unistd.h>
//#include <memory>
#include <algorithm>
#include <vector>


const std::size_t N = 1 << 20;


__device__ // can only be called from within a kernel, not from the host
void vec_inc(float* const c, const std::size_t n)
{
    for (std::size_t i = threadIdx.x + (blockIdx.x * blockDim.x); i < n; i += (blockDim.x * gridDim.x))
    {
        ++ c[i];
    }
}


__host__ // explicitly disallow this to run on the GPU, cannot be launched nor called from kernels
void vec_inc2(float* const c, const std::size_t n)
{
    for (std::size_t i = 0; i < n; ++ i)
    {
        ++ c[i];
    }
}



__global__
void vec_add(float* const c, const float* const a, const float* const b, const std::size_t n)
{
    for (std::size_t i = threadIdx.x + (blockIdx.x * blockDim.x); i < n; i += (blockDim.x * gridDim.x))
    {
        c[i] = a[i] + b[i];
    }

    vec_inc(c, n);
}


int main(void)
{
    // grids and blocks are topologically laid out similar to the problem
    // for 1D arrays, a grid size of (1,1,1) = 1D, and a block size of (N,1,1) = N, would be enough to fully cover the array
    const dim3 grid_size(1, 1, 1);
    const dim3 block_size(1024, 1, 1);

    std::vector<float> h_a(N);
    std::vector<float> h_b(N);
    std::vector<float> h_c(N);
    std::size_t error_count = 0;

    float* d_a = NULL;
    float* d_b = NULL;
    float* d_c = NULL;

    hipStream_t stream;

    std::fill(h_a.begin(), h_a.end(), 1.0);
    std::fill(h_b.begin(), h_b.end(), 2.0);

    if (hipSuccess != hipStreamCreate(&stream))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipMalloc(&d_a, N * sizeof(h_a[0])))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipMalloc(&d_b, N * sizeof(h_b[0])))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipMalloc(&d_c, N * sizeof(h_c[0])))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipMemcpyAsync(d_a, h_a.data(), N * sizeof(h_a[0]), hipMemcpyHostToDevice, stream))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipMemcpyAsync(d_b, h_b.data(), N * sizeof(h_b[0]), hipMemcpyHostToDevice, stream))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    vec_add<<<grid_size, block_size, 0, stream>>>(d_c, d_a, d_b, N);

    // this is slower to execute since we can just +1 in the prior kernel
    // however we want it in a separate function to keep code clean
    // so its not able to be launched like a kernel regularly can
    //vec_inc<<<grid_size, block_size, 0, stream>>>(d_c, N);

    if (hipSuccess != hipMemcpyAsync(h_c.data(), d_c, N * sizeof(h_c[0]), hipMemcpyDeviceToHost, stream))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    if (hipSuccess != hipStreamSynchronize(stream))
    {
        std::cout << __LINE__ << std::endl;
        return -__LINE__;
    }

    // now add 1 to everything again, once h_c has been received from the GPU via d_c
    vec_inc2(h_c.data(), h_c.size());

    for (std::size_t i = 0; i < N; ++i)
    {
        if (h_a[i] + h_b[i] + 2 != h_c[i])
        {
            //std::cout << i << " " << h_c[i] << std::endl;
            ++ error_count;
        }
    }

    std::cout << error_count << " " << 100.0 * (static_cast<double>(error_count) / N) << "% mismatched" << std::endl << std::flush;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipStreamDestroy(stream);
    return 0;
}

