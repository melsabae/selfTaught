
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <unistd.h>
#include <memory>

const std::size_t N = 1 << 20;

__global__
void vec_add(float* const c, const float* const a, const float* const b, const std::size_t n)
{
    // shared memory spaces are block-scoped and for intra-thread communication, 10 was meaningless here
    //__shared__ int shared_memory[10];

    // grids have blocks, blocks have threads
    // thread indices are not unique within a grid, so unique-ify it using block information
    const std::size_t i = threadIdx.x + (blockIdx.x * blockDim.x);
    //threadIdx.y + (blockIdx.y * blockDim.y);
    //threadIdx.z + (blockIdx.z * blockDim.z);

    // sometimes the grid is oversized statically for a problem that has an upper bound, though an instance may be smaller
    // so this guard is used in those cases
    // otherwise if the problem is the same size every time, the grid can be exactly sized, and the guard can be removed
    if (i < n)
    {
        c[i] = a[i] + b[i];
    }

    // barriers
    //__syncthreads(); all threads must hit this point before the kernel can finish
}

int main(void)
{
    // grids and blocks are topologically laid out similar to the problem
    // for 1D arrays, a grid size of (1,1,1) = 1D, and a block size of (N,1,1) = N, would be enough to fully cover the array
    const dim3 grid_size(1, 1, 1);
    const dim3 block_size(N, 1, 1);

    std::unique_ptr<float[]> h_a(new float[N]);
    std::unique_ptr<float[]> h_b(new float[N]);
    std::unique_ptr<float[]> h_c(new float[N]);

    std::size_t error_count = 0;

    float* d_a = NULL;
    float* d_b = NULL;
    float* d_c = NULL;

    hipMalloc(&d_a, N * sizeof(h_a[0]));
    hipMalloc(&d_b, N * sizeof(h_b[0]));
    hipMalloc(&d_c, N * sizeof(h_c[0]));

    for (std::size_t i = 0; i < N; ++ i)
    {
        h_a[i] = 1;
        h_b[i] = 2;
    }

    // for the time being, this code does not work
    hipMemcpy(d_a, h_a.get(), N * sizeof(h_a[0]), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.get(), N * sizeof(h_b[0]), hipMemcpyHostToDevice);
    hipMemset(d_c, 0, N * sizeof(h_c[0]));
    vec_add<<<grid_size, block_size>>>(d_c, d_a, d_b, N);

    hipDeviceSynchronize(); // kernels are launched async, host immediately returns, so block until the computation is done

    // kernels are launched async from the host, but are scheduled serially per device
    // since this machine has a single GPU, they'd be scheduled sequentially, but the host would be operating async

    hipMemcpy(h_c.get(), d_c, sizeof(h_c), hipMemcpyDeviceToHost);

    for (std::size_t i = 0; i < N; ++ i)
    {
        //if ((3 * (i + 1)) != h_c[i])
        std::cout << h_c[i] << std::endl;
        if (3 != h_c[i])
        {
            //std::cout << i << std::endl;
            //++ error_count;
        }
    }

    std::cout << 100.0 * (static_cast<double>(error_count) / N) << "% mismatched" << std::endl << std::flush;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}

