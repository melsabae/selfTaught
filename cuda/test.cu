
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <unistd.h>
#include <memory>
#include <algorithm>
#include <vector>

const std::size_t N = 1 << 20;

__global__
void vec_add(float* const c, const float* const a, const float* const b, const std::size_t n)
{
    for (std::size_t i = threadIdx.x + (blockIdx.x * blockDim.x); i < n; i += (blockDim.x * gridDim.x))
    {
        c[i] = a[i] + b[i];
    }
}

int main(void)
{
    // grids and blocks are topologically laid out similar to the problem
    // for 1D arrays, a grid size of (1,1,1) = 1D, and a block size of (N,1,1) = N, would be enough to fully cover the array
    const dim3 grid_size(1, 1, 1);
    const dim3 block_size(1024, 1, 1);

    std::vector<float> h_a(N);
    std::vector<float> h_b(N);
    std::vector<float> h_c(N);

    std::size_t error_count = 0;

    float* d_a = NULL;
    float* d_b = NULL;
    float* d_c = NULL;

    hipMalloc(&d_a, N * sizeof(h_a[0]));
    hipMalloc(&d_b, N * sizeof(h_b[0]));
    hipMalloc(&d_c, N * sizeof(h_c[0]));

    std::fill(h_a.begin(), h_a.end(), 1.0);
    std::fill(h_b.begin(), h_b.end(), 2.0);

    hipMemcpy(d_a, h_a.data(), N * sizeof(h_a[0]), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.data(), N * sizeof(h_b[0]), hipMemcpyHostToDevice);

    vec_add<<<grid_size, block_size>>>(d_c, d_a, d_b, N);

    hipMemcpy(h_c.data(), d_c, N * sizeof(h_c[0]), hipMemcpyDeviceToHost);
    hipDeviceSynchronize(); // kernels are launched async, host immediately returns, so block until the computation is done

    for (std::size_t i = 0; i < N; ++ i)
    {
        if (h_a[i] + h_b[i] != h_c[i])
        {
            //std::cout << i << " " << h_c[i] << std::endl;
            ++ error_count;
        }
    }

    std::cout << error_count << " " << 100.0 * (static_cast<double>(error_count) / N) << "% mismatched" << std::endl << std::flush;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}

