
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <unistd.h>
#include <memory>

const std::size_t N = 1 << 20;

__global__
void vec_add(float* const c, const float* const a, const float* const b, const std::size_t n)
{
    //__shared__ int shared_memory[10]; // 10 is meaningless, this is just for intra-thread communication

    const std::size_t i = threadIdx.x + (blockIdx.x * blockDim.x);
    //threadIdx.y + (blockIdx.y * blockDim.y);
    //threadIdx.z + (blockIdx.z * blockDim.z);

    if (i < n)
    {
        c[i] = a[i] + b[i];
    }

    // barriers

}

int main(void)
{
    const dim3 grid_size(1, 1, 1);
    const dim3 block_size(N, 1, 1);

    std::unique_ptr<float[]> h_a(new float[N]);
    std::unique_ptr<float[]> h_b(new float[N]);
    std::unique_ptr<float[]> h_c(new float[N]);

    std::size_t error_count = 0;

    float* d_a = NULL;
    float* d_b = NULL;
    float* d_c = NULL;

    hipMalloc(&d_a, N * sizeof(h_a[0]));
    hipMalloc(&d_b, N * sizeof(h_b[0]));
    hipMalloc(&d_c, N * sizeof(h_c[0]));

    for (std::size_t i = 0; i < N; ++ i)
    {
        h_a[i] = 1;
        h_b[i] = 2;
    }

    hipMemcpy(d_a, h_a.get(), N * sizeof(h_a[0]), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.get(), N * sizeof(h_b[0]), hipMemcpyHostToDevice);
    hipMemset(d_c, 0, N * sizeof(h_c[0]));
    vec_add<<<grid_size, block_size>>>(d_c, d_a, d_b, N);
    hipMemcpy(h_c.get(), d_c, sizeof(h_c), hipMemcpyDeviceToHost);

    for (std::size_t i = 0; i < N; ++ i)
    {
        //if ((3 * (i + 1)) != h_c[i])
        if (3 != h_c[i])
        {
            //std::cout << i << std::endl;
            ++ error_count;
        }
    }

    std::cout << 100.0 * (static_cast<double>(error_count) / N) << "% mismatched" << std::endl << std::flush;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}

